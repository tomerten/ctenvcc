#include "hip/hip_runtime.h"
#include "Numeric.cuh"
//#include "Numeric.cu"
#include "Constants.cuh"

namespace COMMON {

    void setBasic(std::map<std::string, double> &twheader, std::map<std::string, double> &paramMap){
        double gamma   = twheader["GAMMA"];
        double gammatr = twheader["GAMMATR"];
        double p0      = twheader["PC"];
        double len     = twheader["LENGTH"];
        double mass    = twheader["MASS"];

        paramMap["betar"]   = NUMERIC::BetaR( gamma );
        paramMap["trev"]    = len / ( paramMap["betar"] * CONSTANTS::clight );
        paramMap["frev"]    = 1.0 / paramMap["trev"];
        paramMap["omega"]   = 2.0 * CONSTANTS::pi * paramMap["frev"];
        paramMap["eta"]     = NUMERIC::eta(gamma, gammatr);
        paramMap["betxavg"] = twheader["LENGTH"] / (twheader["Q1"] * 2.0 * CONSTANTS::pi);
        paramMap["betyavg"] = twheader["LENGTH"] / (twheader["Q2"] * 2.0 * CONSTANTS::pi);
       
    }

    void setLongParam(
        std::map<std::string, double> &twheader,
        std::map<std::string, double> &paramMap,
        std::map<std::string, double> &inputMapDouble,
        std::map<std::string, std::vector<double>> &inputMapVector,
        std::map<int, std::vector<double>> &bunchMap
    ){
        double gamma    = twheader["GAMMA"];
        double gammatr  = twheader["GAMMATR"];
        double h0       = inputMapVector["HarmonicNumbers"][0];
        double U0       = paramMap["U0"];
        double charge   = paramMap["charge"];
        double omega    = paramMap["omega"];
        double betar    = paramMap["betar"];
        double angularf = 2.0 * CONSTANTS::pi * h0;
        std::vector<double> h = inputMapVector["HarmonicNumbers"];
        std::vector<double> v = inputMapVector["Voltages"];

        std::for_each( bunchMap.begin(), bunchMap.end(),
            [&](std::pair<const int, std::vector<double>> &p){
                // defining phis search parameters
                double search = (double)p.first * 2.0 * CONSTANTS::pi + angularf /
                    ( 8.0 * *std::max_element(h.begin(), h.end()) );
                double searchWidth =  angularf / (2.0 * *std::max_element(h.begin(), h.end()));

                double eps = 1e-6;
                double phis     = NUMERIC::SynchronuousPhase( 0.0, search, U0, charge, v, h, eps  );
                double phisNext = NUMERIC::SynchronuousPhase( 0.0, search + CONSTANTS::pi, U0, charge, v, h, eps  );
                double qs       = NUMERIC::SynchrotronTune( phis, twheader, paramMap, v, h);
                double tauhat   = fabs( phisNext - phis ) / ( h0 * omega );
                double sigs     = p.second[3];
                double sige     = NUMERIC::sigefromsigs( omega, sigs, qs, gamma, gammatr ) ;
                double dpop     = NUMERIC::dee_to_dpp( sige, betar );

                p.second.push_back(phis);
                p.second.push_back(phisNext);
                p.second.push_back(qs);
                p.second.push_back(tauhat);
                p.second.push_back(sige);
                p.second.push_back(dpop);
            }
    );
        
    

    }
}