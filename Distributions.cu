#include "hip/hip_runtime.h"
#include "Random.cuh"
#include "Random.cu"
#include "Hamiltonian.cuh"
#include "Hamiltonian.cu"
#include <vector>
#include <thrust/host_vector.h>
#include <thrust/transform.h>
#include <thrust/sequence.h>
#include <thrust/copy.h>
#include <thrust/fill.h>
#include <thrust/replace.h>
#include <thrust/functional.h>
#include <map>

namespace DISTRIBUTIONS{

  std::vector<double> BiGaussian4D(
    std::map<std::string, double> &paramMap,
    std::vector<double> &bunchMapRow,
    int seed) {
        std::vector<double> out;
      
        static double ampx, ampy, amp, r1, r2, facc;
        static double x, px, y, py;
        double betax = paramMap["betxavg"] ;
        double betay = paramMap["betyavg"];
        double ex    = bunchMapRow[1];
        double ey    = bunchMapRow[2];
        // 1 sigma rms beam sizes using average ring betas
        ampx = sqrt(betax * ex);
        ampy = sqrt(betay * ey);

        // generate bi-gaussian distribution in the x-px phase-space
        do {
            r1 = 2 * RANDOM::ran3(&seed) - 1;
            r2 = 2 * RANDOM::ran3(&seed) - 1;
            amp = r1 * r1 + r2 * r2;
        } while (amp >= 1);

        facc =
            sqrt(-2 * log(amp) /
                amp); // transforming [-1,1] uniform to gaussian - inverse transform

        x = ampx * r1 * facc;  // scaling the gaussian
        px = ampx * r2 * facc; // scaling the gaussian

        // generate bi-gaussian distribution in the y-py phase-space
        do {
            r1 = 2 * RANDOM::ran3(&seed) - 1;
            r2 = 2 * RANDOM::ran3(&seed) - 1;
            amp = r1 * r1 + r2 * r2;
        } while (amp >= 1);

        // transforming [-1,1] uniform to gaussian - inverse transform
        facc = sqrt(-2 * log(amp) /amp); 

        y  = ampy * r1 * facc; // scaling the gaussian
        py = ampy * r2 * facc; // scaling the gaussian

        out.push_back(x);
        out.push_back(px);
        out.push_back(y);
        out.push_back(py);

        return out;
    }
  
    std::vector<double> BiGaussian6DLongMatched(
        std::map<std::string, double> &paramMap,
        std::map<std::string, std::vector<double>> &inputMapVector,
        std::map<std::string, double> &twheader,
        std::vector<double> &bunchMapRow,
        int seed) {

            std::vector<double> h = inputMapVector["HarmonicNumbers"];

            double h0      = h[0];
            double tauhat  = bunchMapRow[7];
            double omega   = paramMap["omega"];
            double ampt    = bunchMapRow[3] / CONSTANTS::clight;
            double neta    = paramMap["eta"];
            double charge  = paramMap["charge"];
            double betar   = paramMap["betar"];
            double p0      = twheader["PC"];

            // Max value Hamiltonian that is stable
            // is, with the sign convention used, left of the ham contour
            // at 180-phis (The Ham rises lin to the right.)
            double hom     = (h0 * omega);
            double ts      = bunchMapRow[4] / hom;
            // int npi        = int(longparam["phis"] / (2.0 * pi));
            // double tperiod = 2.0 * pi / hom;
            // double ts2 = longparam["phisNext"] / hom + double(npi) * tperiod;
            double ts2     = bunchMapRow[5] / hom ;// + double(npi) * tperiod;
            double delta   = bunchMapRow[9];

            std::vector<double> out;
            out = BiGaussian4D(paramMap, bunchMapRow, seed);

            // adding two zeros
            out.push_back(0.0);
            out.push_back(0.0);

            double r1, r2, amp, facc, tc, ham, hammin;
            tc = (omega * neta * h0);
            //pc = (omega * charge ) / (2.0 * CONSTANTS::pi * p0 * 1.0e9 * betar);

            // max Hamiltonian (phis is the unstable point here - bunchMapRow[4])
            // as we want the max value for the Hamiltonian -> stable is around
            // the minimum.
            double hammax = HAMILTONIAN::Hamiltonian(
                twheader, 
                paramMap, 
                inputMapVector,
                bunchMapRow[4],
                tc, ts, 0.0);
  
                // select valid t values
                do {
                    r1 = 2.0 * RANDOM::ran3(&seed) - 1.0;
                    r2 = 2.0 * RANDOM::ran3(&seed) - 1.0;
                    amp = r1 * r1 + r2 * r2;
                    if (amp >= 1)
                        continue;

                    facc = sqrt(-2 * log(amp) / amp);
                    out[4] = ts2 + ampt * r1 * facc;
    
                    if (abs(out[4] - ts2) >= abs(ts - ts2))
                        continue;

                    // min Hamiltonian
                    hammin = HAMILTONIAN::Hamiltonian(
                        twheader, 
                        paramMap, 
                        inputMapVector,
                        bunchMapRow[4],
                        tc, out[4], 0.0);

                } while ((hammin > hammax) || (abs(out[4] - ts2) >= abs(ts - ts2)));

                // select matched deltas
                do {
                    r1 = 2.0 * RANDOM::ran3(&seed) - 1;
                    r2 = 2.0 * RANDOM::ran3(&seed) - 1;
                    amp = r1 * r1 + r2 * r2;

                    if (amp >= 1)
                    continue;

                    facc = sqrt(-2.0 * log(amp) / amp);
                    out[5] = bunchMapRow[9] * r2 * facc;

                    ham = HAMILTONIAN::Hamiltonian(
                        twheader, 
                        paramMap, 
                        inputMapVector,
                        bunchMapRow[4],
                        tc, out[4],out[5]);
                } while ((ham < hammin) || (ham > hammax));

            return out;
    }
    
    std::vector<std::vector<double>> GenerateDistributionMatched(
        std::map<std::string, double> &paramMap,
        std::map<std::string, std::vector<double>> &inputMapVector,
        std::map<std::string, int> &inputMapInt,
        std::map<std::string, double> &twheader,
        std::vector<double> &bunchMapRow
       ) {
      std::vector<std::vector<double>> out;
      int nMacro = inputMapInt["nMacro"];
      int seed = inputMapInt["seed"];
      for (int i = 0; i < nMacro; i++) {
        out.push_back(BiGaussian6DLongMatched(paramMap, inputMapVector, twheader, bunchMapRow, seed));
      }
      return out;
    }
    

}