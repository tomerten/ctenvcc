#include "hip/hip_runtime.h"
#include "Random.cuh"
#include "Random.cu"
#include <vector>
#include <thrust/host_vector.h>
#include <map>

namespace DISTRIBUTIONS{

  thrust::host_vector<double> BiGaussian4D(
    std::map<std::string, double> &paramMap,
    std::vector<double> &bunchMapRow,
    int seed) {
        thrust::host_vector<double> out;
      
        static double ampx, ampy, amp, r1, r2, facc;
        static double x, px, y, py;
        double betax = paramMap["betxavg"] ;
        double betay = paramMap["betyavg"];
        double ex    = bunchMapRow[1];
        double ey    = bunchMapRow[2];
        // 1 sigma rms beam sizes using average ring betas
        ampx = sqrt(betax * ex);
        ampy = sqrt(betay * ey);

        // generate bi-gaussian distribution in the x-px phase-space
        do {
            r1 = 2 * RANDOM::ran3(&seed) - 1;
            r2 = 2 * RANDOM::ran3(&seed) - 1;
            amp = r1 * r1 + r2 * r2;
        } while (amp >= 1);

        facc =
            sqrt(-2 * log(amp) /
                amp); // transforming [-1,1] uniform to gaussian - inverse transform

        x = ampx * r1 * facc;  // scaling the gaussian
        px = ampx * r2 * facc; // scaling the gaussian

        // generate bi-gaussian distribution in the y-py phase-space
        do {
            r1 = 2 * RANDOM::ran3(&seed) - 1;
            r2 = 2 * RANDOM::ran3(&seed) - 1;
            amp = r1 * r1 + r2 * r2;
        } while (amp >= 1);

        // transforming [-1,1] uniform to gaussian - inverse transform
        facc = sqrt(-2 * log(amp) /amp); 

        y  = ampy * r1 * facc; // scaling the gaussian
        py = ampy * r2 * facc; // scaling the gaussian

        out.push_back(x);
        out.push_back(px);
        out.push_back(y);
        out.push_back(py);

        return out;
    }
    
}