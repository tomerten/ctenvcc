#include "hip/hip_runtime.h"
namespace HAMILTONIAN {

    double tcoeff(
        std::map<std::string, double> &paramMap, 
        double h0) {
        
            return paramMap["omega"] * paramMap["eta"] * h0;
        }

    double pcoeff(
        std::map<std::string, double> &paramMap, 
        std::map<std::string, double> &twheader, 
        double voltage) {
            // factor 1.0e9 -> pc is in GeV
            return paramMap["omega"] * voltage * paramMap["charge"] /
                   (2.0 * CONSTANTS::pi * twheader["PC"] * 1.0e9 * paramMap["betar"]);
        }


    double Hamiltonian(
        std::map<std::string, double> &twheader,
        std::map<std::string, double> &paramMap,
        std::map<std::string, std::vector<double>> &inputMapVector,
        double phis,
        double tcoeff, 
        double t, 
        double delta) {
            double kinetic, potential;
            std::vector<double> h = inputMapVector["HarmonicNumbers"];
            std::vector<double> v = inputMapVector["Voltages"];

            // kinetic contribution
            // We assume initial bunch length is given
            kinetic = 0.5 * tcoeff * delta * delta;

            std::vector<double> pcoeffs, hRatios, hRatiosInv, phases;

            // calculate coefficients for the determining the potential
            for (int i = 0; i < h.size(); i++) {
                pcoeffs.push_back( pcoeff( paramMap, twheader, v[i] ) );
                phases.push_back( h[i] * paramMap["omega"] * t );
                hRatios.push_back( h[0] / h[i] );
                hRatiosInv.push_back( h[i] / h[0] );
            }

            // calc the potential
            potential = pcoeffs[0] * (cos(phases[0]) - cos(phis) +
                 (phases[0] - phis) * sin(phis));

            for (int i = 1; i < h.size(); i++) {
                potential += pcoeffs[i] * hRatios[i] *
                    (cos(phases[i]) - cos(hRatiosInv[i] * phis) +
                    (phases[i] - hRatiosInv[i] * phis) * sin(hRatiosInv[i] * phis));
            }

            return kinetic + potential;
        }




}