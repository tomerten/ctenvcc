
#include <hip/hip_runtime.h>
#include <algorithm>
#include <cstdlib>
#include <fstream>
#include <iostream>
#include <iterator>
#include <list>
#include <map>
#include <sstream>
#include <stdio.h>
#include <string>
#include <unordered_map>
#include <vector>

using namespace std;

map<string, double> GetTwissHeader(string filename) {
  vector<string> TWISSHEADERKEYS /* */ {
      "MASS",     "CHARGE",  "ENERGY",  "PC",      "GAMMA",   "KBUNCH",
      "BCURRENT", "SIGE",    "SIGT",    "NPART",   "EX",      "EY",
      "ET",       "BV_FLAG", "LENGTH",  "ALFA",    "ORBIT5",  "GAMMATR",
      "Q1",       "Q2",      "DQ1",     "DQ2",     "DXMAX",   "DYMAX",
      "XCOMAX",   "YCOMAX",  "BETXMAX", "BETYMAX", "XCORMS",  "YCORMS",
      "DXRMS",    "DYRMS",   "DELTAP",  "SYNCH_1", "SYNCH_2", "SYNCH_3",
      "SYNCH_4",  "SYNCH_5"};
  map<string, double> out;
  string line;
  ifstream file(filename);

  vector<int> labels;
  map<int, int> colmap;

  getline(file, line);
  // check if file is open
  if (file.is_open()) {
    // printf("File is open. Reading Twiss header.\n");
    // read lines until eof
    int counter = 0;
    string key;
    string at;
    double value;
    while (!file.eof() && counter < 41) {
      counter++;
      getline(file, line);
      istringstream iss(line);
      iss >> at >> key >> at >> value;
      vector<string>::iterator it =
          find(TWISSHEADERKEYS.begin(), TWISSHEADERKEYS.end(), key);
      // cout << key << " " << value << " " << endl;
      if (it != TWISSHEADERKEYS.end()) {
        out[key] = value;
      }
    }
  }
  file.close();
  // printf("File is closed. Done reading Twiss header.\n");
  return out;
}


map<string, vector<double>> GetTwissTableAsMap(string filename) {
  vector<string> TWISSCOLS /* */ {"L",    "BETX",  "ALFX", "BETY", "ALFY",
                                  "DX",   "DPX",   "DY",   "DPY",  "K1L",
                                  "K1SL", "ANGLE", "K2L",  "K2SL"};
  map<string, vector<double>> out;
  map<int, string> columnnames;

  string line;
  ifstream file(filename);

  if (file.is_open()) {
    // printf("File is open\n");

    vector<double> row;
    int rowcounter = 0;
    while (!file.eof()) {
      // update row counter
      rowcounter++;

      // read a line
      getline(file, line);

      // if line is 47 read the column names
      if (rowcounter == 47) {
        // load the current line as stream
        istringstream iss(line);

        // split the line and save in vector
        // vector<string> labels;
        int colcounter = 0;
        // cout << "Col idx: ";
        do {
          string sub;
          iss >> sub;
          vector<string>::iterator it =
              find(TWISSCOLS.begin(), TWISSCOLS.end(), sub);
          if (it != TWISSCOLS.end()) {
            columnnames[colcounter - 1] = sub;
            // cout << colcounter - 1 << " " << sub << endl;
          }
          ++colcounter;
        } while (iss);
      }
      if (rowcounter > 48) {
        istringstream iss(line);
        int colcounter = 0;
        do {
          string sub;
          string key;
          double value;
          iss >> sub;
          if (columnnames.count(colcounter) > 0) {
            key = columnnames[colcounter];
            value = stod(sub);
            // cout << key << " " << value << endl;
            out[key].push_back(value);
          }
          ++colcounter;
        } while (iss);
        // cout << endl;
      }
    }
  }
  file.close();
  return out;
}
