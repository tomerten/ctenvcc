#include "hip/hip_runtime.h"
#include <math.h>
#include "Constants.cuh"

namespace NUMERIC{
    double BetaR( double gamma ){
        return sqrt( 1.0 - ( 1.0 / ( gamma * gamma )) );
    }

    double ParticleRadius( double charge, double aatom ){
        return charge * charge * CONSTANTS::prad / aatom; 
    }

    double eta( double gamma, double gammatr ) {
        return 1.0 / ( gammatr * gammatr - 1.0 / ( gamma * gamma ) );
      }

    double VoltageRfeV( double phi, std::vector<double> &volts, std::vector<double> &hs, double charge ) {
        double vrf = volts[0] * sin(phi);

        for (int i = 1; i < hs.size(); i++) {
            vrf += volts[i] * sin((hs[i] / hs[0]) * phi);
        }
        
        // V -> eV
        vrf *= charge;

        return vrf;
    }

    double VoltageRfeVPrime( double phi, std::vector<double> &volts,std::vector<double> &hs, double charge ) {
        // init - phi is in rad
        double vrf = volts[0] * cos(phi);

        // add other rfs
        for (int i = 1; i < volts.size(); i++) {
        vrf += volts[i] * (hs[i] / hs[0]) * cos((hs[i] / hs[0]) * phi);
        }

        // V -> eV
        vrf *= charge;

        return vrf;
    }

    double SynchronuousPhase(
        double target, double init_phi, double U0,double charge, 
        std::vector<double> &volts,
        std::vector<double> &hs, 
        double epsilon) {

        // Set the initial option prices and volatility
        double y = VoltageRfeV(init_phi, volts, hs, charge) - U0;
        double x = init_phi;

        // Newton Raphson
        // REF: https://www.boost.org/doc/libs/1_62_0/libs/math/doc/html/math_toolkit/roots/roots_deriv.html#math_toolkit.roots.roots_deriv.newton
        // x_{n+1}= x_n - \frac{f(x)}{f'(x)}
        while (fabs(y - target) > epsilon) {
            double d_x = VoltageRfeVPrime(x, volts, hs, charge);
            x += (target - y) / d_x;
            y = VoltageRfeV(x, volts, hs, charge) - U0;
        }

        return x;
    }

    double SynchrotronTune(
        double phis,
        std::map<std::string, double> &twheader,
        std::map<std::string, double> &paramMap,
        std::vector<double> &volts, 
        std::vector<double> &hs) {

            double charge = paramMap["charge"];
            double pc     = twheader["PC"];
            double n      = paramMap["eta"];

            return sqrt(hs[0] * n *
                fabs( charge * VoltageRfeVPrime(phis, volts, hs, charge) ) / ( 2.0 * CONSTANTS::pi * pc * 1.0e9 ) );
    }

    double sigefromsigs(double omega0, double sigs, double qs, double gamma, double gammatr) {
        // dE/E = Beta**2 dp/p
        double beta2 = BetaR(gamma);
        beta2 *= beta2;
        return beta2 * qs * omega0 * (sigs / (fabs(eta(gamma, gammatr)) * CONSTANTS::clight));
    }

    double dee_to_dpp(double dee, double beta0) {
        return sqrt(((dee + 1.0) * (dee + 1.0) - 1.0) / (beta0 * beta0) + 1.0) - 1.0;
    }

}