#include "hip/hip_runtime.h"
#include "Constants.cuh"
#include "Numeric.cuh"
#include "Numeric.cu"

namespace RADIATION{
    double RadiationLossesPerTurn( 
        std::map<std::string, double> &twiss,
        std::map<std::string, double> &beamParMap  
        ) {
        double gamma  = twiss["GAMMA"];
        double p0     = twiss["PC"];
        double len    = twiss["LENGTH"];
        double mass   = twiss["MASS"];
        double I2     = twiss["SYNCH_2"];
      
        // beam dependent vars - there can be different particles in the different beams
        double aatom  = beamParMap["aatom"];
        double trev   = beamParMap["trev"];
        double charge = beamParMap["charge"];

        // REF: Handbook for accelerator physicists and engineers - sec edition
        double particle_radius = NUMERIC::ParticleRadius( charge, aatom );
        double cgamma = (4.0 * CONSTANTS::pi / 3.0) * ( particle_radius / ( mass * mass * mass ) );
      
        return (CONSTANTS::clight * cgamma) / (2.0 * CONSTANTS::pi * len) * p0 * p0 * p0 * p0 * I2 * 1.0e9 *
               trev;
    }

    void RadEquilib(
        std::map<std::string, double> &twheader,
        std::map<std::string, double> &paramMap) {

        // base quantities
        double gamma   = twheader["GAMMA"];
        double gammatr = twheader["GAMMATR"];
        double p0      = twheader["PC"] * 1.0e9;
        double len     = twheader["LENGTH"] * 1.0;
        double restE   = twheader["MASS"] * 1.0e9;
        double charge  = paramMap["charge"];
        double aatom   = paramMap["aatom"];
        double qs      = paramMap["qs"];
        double omega   = paramMap["omega"];
        double q1      = twheader["Q1"];

        // use madx rad integrals
        double I1  = twheader["SYNCH_1"];
        double I2  = twheader["SYNCH_2"];
        double I3  = twheader["SYNCH_3"];
        double I4x = twheader["SYNCH_4"];
        double I5x = twheader["SYNCH_5"];

        double I4y = 0.0;
        double I5y = 0.0;

        // derived quantities
        double pradius = NUMERIC::ParticleRadius(charge, aatom);
        double CalphaEC = pradius * CONSTANTS::clight / (3.0 * restE * restE * restE) * (p0 * p0 * p0 / len);

        // transverse partition numbers
        double jx = 1.0 - I4x / I2;
        double jy = 1.0 - I4y / I2;
        double alphax = 2.0 * CalphaEC * I2 * jx;
        double alphay = 2.0 * CalphaEC * I2 * jy;
        double alphas = 2.0 * CalphaEC * I2 * (jx + jy);

        // mc**2 expressed in Joule to match units of cq
        double mass    = restE * CONSTANTS::electron_volt_joule_relationship;
        double cq      = 55.0 / (32.0 * sqrt(3.0)) * (CONSTANTS::hbarsu * CONSTANTS::clight) / mass;
        double sigE0E2 = cq * gamma * gamma * I3 / (2.0 * I2 + I4x + I4y);

        // ! = deltaE/E_0 see wiedemann p. 302,
        // and Wolski: E/(p0*c) - 1/beta0 = (E - E0)/(p0*c) = \Delta E/E0*beta0 with
        // E0 = p0*c/beta0 therefore:
        double betar = NUMERIC::BetaR(gamma);
        double dpop  = NUMERIC::dee_to_dpp(sqrt(sigE0E2), betar);
        double sigs  = dpop * len * NUMERIC::eta(gamma, gammatr) / (2 * CONSTANTS::pi * qs);
        double exinf = cq * gamma * gamma * I5x / (jx * I2);
        double eyinf = cq * gamma * gamma * I5y / (jy * I2);
        
        double betaAvg = paramMap["betxavg"];

        eyinf = (eyinf == 0.0) ? cq * betaAvg * I3 / (2.0 * jy * I2) : eyinf;

        paramMap["taux"] = 1.0 / alphax;
        paramMap["tauy"] = 1.0 / alphay;
        paramMap["taus"] = 1.0 / alphas;
        paramMap["exinf"] = exinf;
        paramMap["eyinf"] = eyinf;
        paramMap["sigeoe2"] = sigE0E2;
        paramMap["sigsinf"] = sigs;
        paramMap["jx"] = jx;
        paramMap["jy"] = jy;
    }


    void RadDecayExcitationCoeff(
        std::map<std::string, double> &twheader,
        std::map<std::string, double> &paramMap) {

        double gamma     = twheader["GAMMA"];
        double trev      = paramMap["trev"];
        double timeratio = paramMap["timeratio"];
        double taus      = paramMap["taus"];
        double taux      = paramMap["taux"];
        double tauy      = paramMap["tauy"];
        double sigsinf   = paramMap["sigsinf"];
        double tt        = trev * timeratio;

        double sigx = sqrt(paramMap["exinf"] * paramMap["betxavg"]);
        double sigy = sqrt(paramMap["eyinf"] * paramMap["betyavg"]);

        // timeratio is real machine turns over per simulation turn
        paramMap["coeffdecaylong"] = exp(-tt / taus);

        // excitation uses a uniform distibution on [-1:1]
        // sqrt(3) * sigma => +/-3 sigma**2
        // see also lecture 2 Wolski on linear dynamics and radiation damping
        // not sure about the sqrt3
        paramMap["coeffexcitelong"] = (paramMap["sigeoe2"] * gamma) / sqrt(3) *
                    sqrt(3.) * sqrt(1.0 * tt / taus);

        // the damping time is for EMITTANCE, therefore need to multiply by 2
        paramMap["coeffdecayx"] = exp(-(tt / (2 * taux)));
        paramMap["coeffdecayy"] = exp(-(tt / (2 * tauy)));

        // exact:
        // coeffgrow= sigperp*sqrt(3.)*sqrt(1-coeffdecay**2)
        // squared because sigma and not emit
        paramMap["coeffgrowx"] = sigx * sqrt(3.) * sqrt(1.0 - pow(paramMap["coeffdecayx"], 2));
        paramMap["coeffgrowy"] = sigy * sqrt(3.) * sqrt(1.0 - pow(paramMap["coeffdecayy"], 2));
    }
}